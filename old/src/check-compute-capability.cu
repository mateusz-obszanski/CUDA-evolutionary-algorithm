
#include <hip/hip_runtime.h>
#include <iostream>

int
main() {
    hipDeviceProp_t props;
    const int      device = 0;
    hipGetDeviceProperties(&props, device);

    std::cout << "version: " << props.major << '.' << props.minor << '\n';
}
