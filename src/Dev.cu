#include "hip/hip_runtime.h"
#include "device/errors.cuh"
#include "device/memory/allocator.cuh"
#include "device/memory/memory.cuh"
#include "device/random.cuh"
#include "device/reordering.cuh"
#include <cuda/std/cstddef>
#include <cuda/std/iterator>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

__global__ void
mul2(int* out, cuda::std::size_t len) {
    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= len)
        return;

    out[tid] *= 2;
}

template <typename Iter>
__global__ void
iterPrintDevice(Iter begin, Iter end) {
    const auto diff = end - begin;

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= std::abs(diff))
        return;

    const auto p = begin + tid;

    printf("printf from device: %d\n", *p);
}

template <typename IterT>
inline void
printIter(IterT begin, IterT end) {
    std::cout << '[';
    thrust::copy(
        begin, end,
        std::ostream_iterator<typename IterT::value_type>(std::cout, ", "));
    std::cout << "]\n";
}

template <typename V>
inline void
printVec(const V& v) {
    printIter(v.begin(), v.end());
}

template <typename IterT>
inline void
PrintVecN(const IterT begin, std::size_t n) {
    auto end = begin;
    thrust::advance(end, n);
    printIter(begin, end);
}

void
testManagedMemory() {
    constexpr cuda::std::size_t N = 5;

    int* arr;

    hipMallocManaged(&arr, N * sizeof(int));

    for (size_t i{0}; i < N; ++i)
        arr[i] = i;

    mul2<<<1, N>>>(arr, N);
    if (const auto err = hipGetLastError())
        std::cout << "GPU ERROR: " << hipGetErrorName(err) << ": " << hipGetErrorString(err) << '\n';

    thrust::copy(arr, arr + N, std::ostream_iterator<int>(std::cout, ", "));
    std::cout << '\n';

    hipFree(arr);
}

void
testDeviceMemoryClass() {
    device::memory::raii::Memory<int> mem(4);
    iterPrintDevice<<<1, mem.size()>>>(mem.crbegin(), mem.crend());
    // vvv SEGFAULT
    // thrust::fill(mem.begin(), mem.end(), 42);
    thrust::fill(mem.begin_thrust(), mem.end_thrust(), 42);
    std::cout << mem.rbegin_thrust().base() << '\n';
    // or
    thrust::fill(mem.rbegin_thrust(), mem.rend_thrust(), 2137);

    auto newMem = mem.copy<float>();

    iterPrintDevice<<<1, mem.size()>>>(mem.cbegin(), mem.cend());

    mem.print();
    newMem.print();
}

template <typename State>
__global__ void
skipahead_kernel(unsigned long long n, State* state) {
    // for cuRAND
    curand_skipahead(n, state);
}

template <typename DevIter>
inline thrust::device_vector<typename DevIter::value_type>
makeCopy(DevIter begin, DevIter end) {
    thrust::device_vector<typename DevIter::value_type> xs(thrust::distance(begin, end));
    thrust::copy(begin, end, xs.begin());

    return xs;
}

template <typename DevIter>
inline thrust::device_vector<typename DevIter::value_type>
makeCopyN(DevIter begin, std::size_t n) {
    thrust::device_vector<typename DevIter::value_type> xs(n);
    thrust::copy_n(begin, n, xs.begin());

    return xs;
}

template <typename T = int>
inline thrust::device_vector<T>
makeSequence(std::size_t n) {
    thrust::device_vector<T> result(n);
    thrust::sequence(result.begin(), result.end());

    return result;
}

void
testRnd() {
    constexpr int                    N = 64;
    device::random::RndStateMemory<> states(N);
    device::random::initialize_rnd_states(states);
    device::memory::raii::DeviceMemory<float> random_numbers(N);
    device::random::uniform(random_numbers.begin(), random_numbers.end(), states);

    hipDeviceSynchronize();
    states.print();
    random_numbers.print();
}

void
testRndMask() {
    device::random::RndStateMemory<> states(64);
    device::random::initialize_rnd_states(states);
    thrust::device_vector<bool> mask(states.size());

    device::random::mask(mask.begin(), mask.end(), states, 0.5);
    hipDeviceSynchronize();

    printVec(mask);
}

void
testShuffleMasked() {
    device::random::RndStateMemory<> states(10);
    device::random::initialize_rnd_states(states);
    thrust::default_random_engine rng(0);

    thrust::device_vector<bool> mask(states.size());
    device::random::mask(mask.begin(), mask.end(), states, 0.3);
    hipDeviceSynchronize();

    std::cout << "mask:   ";
    printVec(mask);

    thrust::device_vector<int> seq(mask.size());
    thrust::sequence(seq.begin(), seq.end());

    std::cout << "before: ";
    printVec(seq);

    device::random::shuffle_masked(seq.begin(), seq.end(), mask.begin(), rng);
    hipDeviceSynchronize();

    std::cout << "after:  ";
    printVec(seq);
}

void
testShuffleWithProbability() {
    std::cout << "testShuffleWithProbability\n";

    device::random::RndStateMemory<> states(10);
    device::random::initialize_rnd_states(states);
    thrust::default_random_engine rng(0);

    thrust::device_vector<int> seq(states.size());
    thrust::sequence(seq.begin(), seq.end());

    std::cout << "before: ";
    printVec(seq);

    device::random::shuffle_with_prob(seq.begin(), seq.end(), 0.3, states, rng);

    std::cout << "after:  ";
    printVec(seq);
}

void
testChooseKWithourReplacement() {
    std::cout << "testChooseKWithourReplacement\n";

    device::random::RndStateMemory<> states(10);
    device::random::initialize_rnd_states(states);

    thrust::device_vector<int> seq(states.size());
    thrust::sequence(seq.begin(), seq.end());

    const int k = 3;

    thrust::device_vector<int> choices(k);

    device::random::choose_k_without_replacement(
        seq.begin(), seq.end(), choices.begin(), k, states);

    std::cout << "choices: ";
    printVec(choices);
}

int
main() {
    try {
        // testShuffleMasked();
        testShuffleWithProbability();
        testChooseKWithourReplacement();
        // testRnd();
        // testRndMask();
    } catch (const std::exception& e) {
        std::cout << "ERROR: " << e.what() << '\n';
        return 1;
    }

    return 0;
}
